#include "hip/hip_runtime.h"
﻿#include "lossFunctionsCuda.cuh"
#include <vector>
#include <math.h>
#include <cmath>


float  KullbackLeiblerDivergenceCuda(float* roHat, float& ro, size_t start, size_t end)
{
	float sum = 0.0;
	for (size_t i = start; i < end; i++)
		sum += ro * log(ro / roHat[i]) + (1 - ro) * log((1 - ro) / (1 - roHat[i]));
	return sum;
}

float  KullbackLeiblerDivergenceDerivativeCuda(float& output, float& target)
{
	//TODO არ მუშაობს
	return log(output / target) + 1 / target;
}

float  BinaryCrossentropyCuda(float* output, float* target, size_t targetSize)
{
	float sum = 0;
	for (size_t i = 0; i < targetSize; i++)
	{
		sum += target[i] * log(output[i]) - (1 - target[i]) * log(1 - output[i]);
	}
	return -sum / targetSize;
}

float  BinaryCrossentropyDerivativeCuda(float& output, float& target, size_t size)
{
	return (-target / output + (1 - target) / (1 - output)) / size;
}

float  _CELCuda(float& output, float& target)
{
	return -target * log(output) - (1 - target) * log(1 - output);
}
float CELCuda(float* output, float* target, size_t size)
{
	float sum = 0;
	for (size_t i = 0; i < size; i++)
	{
		sum += _CELCuda(output[i], target[i]);
	}
	return sum / size;
}
float MSLCuda(float& output, float& target)
{
	return pow((target - output), 2) / 2;
}

float MSLCuda(float* output, float* target, size_t start, size_t end, size_t outputSize)
{
	float Sum = 0;
	for (size_t i = start; i < end; i++)
	{
		Sum += MSLCuda(target[i], output[i]) / outputSize;
	}
	return Sum;
}

float CELDerevativeCuda(float& output, float& target)
{
	return -target / output + (1 - target) / (1 - output);
}

float CalculateLossFunctionCuda(int& function, float* output, float* target, size_t start, size_t end, size_t outputSize)
{
	switch (function)
	{
	case static_cast<int>(NeuralEnums::LossFunctionType::MeanSquaredError): return MSLCuda(output, target, start, end, outputSize);
	case static_cast<int>(NeuralEnums::LossFunctionType::BinaryCrossentropy): return BinaryCrossentropyCuda(output, target, outputSize);
		//case NeuralEnums::LossFunctionType::KullbackLeiblerDivergence: return KullbackLeiblerDivergence(output, target);
	default:
		break;
	}
}
float DifferentiateLossWithCuda(float& output, float& target, int& function, size_t size)
{
	switch (function)
	{
	case static_cast<int>(NeuralEnums::LossFunctionType::MeanSquaredError): return output - target;
	case static_cast<int>(NeuralEnums::LossFunctionType::BinaryCrossentropy): return BinaryCrossentropyDerivativeCuda(output, target, size);
	case static_cast<int>(NeuralEnums::LossFunctionType::KullbackLeiblerDivergence): return KullbackLeiblerDivergenceDerivativeCuda(output, target);
	default:
		break;
	}
}
