#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
class CudaClass
{
public:
	int* data;
	int val;
	CudaClass() {}
	CudaClass(int x, int val_) 
	{
		data = new int[1]; data[0] = x;
		val = val_;
	}
};
__device__  CudaClass* cls;
 __global__ void useClass(CudaClass* cudaClass)
{
	cls = cudaClass;
	cls->val = 3;
};
int copyClass()
{
	CudaClass c(2, 2);
	// create class storage on device and copy top level class
	CudaClass* d_c;
	hipMalloc((void**)&d_c, sizeof(CudaClass));
	hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice);
	// make an allocated region on device for use by pointer in class
	int* hostdata;
	hipMalloc((void**)&hostdata, sizeof(int));
	hipMemcpy(hostdata, c.data, sizeof(int), hipMemcpyHostToDevice);
	// copy pointer to allocated device storage to device class
	hipMemcpy(&(d_c->data), &hostdata, sizeof(int*), hipMemcpyHostToDevice);
	useClass << <1, 1 >> > (d_c);
	hipDeviceSynchronize();
	return 0;
}